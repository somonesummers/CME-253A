#include "hip/hip_runtime.h"
// wave 2D GPU
// compile: nvcc -arch=sm_70 -O3 elastic_2D_max.cu
// run: ./a.out
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"

#define USE_SINGLE_PRECISION    /* Comment this line using "//" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT     float
#define PRECIS  4
#else
#define DAT     double
#define PRECIS  8
#endif

#define GPU_ID        3
#define OVERLENGTH_X  1
#define OVERLENGTH_Y  1

#define zeros(A,nx,ny)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc(((nx)*(ny))*sizeof(DAT)); \
                        for(i=0; i < ((nx)*(ny)); i++){ A##_h[i]=(DAT)0.0; }              \
                        hipMalloc(&A##_d      ,((nx)*(ny))*sizeof(DAT));                 \
                        hipMemcpy( A##_d,A##_h,((nx)*(ny))*sizeof(DAT),hipMemcpyHostToDevice);
#define free_all(A)     free(A##_h); hipFree(A##_d);
#define gather(A,nx,ny) hipMemcpy( A##_h,A##_d,((nx)*(ny))*sizeof(DAT),hipMemcpyDeviceToHost);

#define for_ix          int ix = blockIdx.x*blockDim.x + threadIdx.x;
#define for_iy          int iy = blockIdx.y*blockDim.y + threadIdx.y;                              
#define Pres(ix,iy)     (  P[ix + (iy)*nx    ]) 
#define Velx(ix,iy)     ( Vx[ix + (iy)*(nx+1)])        
#define Vely(ix,iy)     ( Vy[ix + (iy)*nx    ]) 
#define t_xx(ix,iy)     (txx[ix + (iy)*nx    ]) 
#define t_yy(ix,iy)     (tyy[ix + (iy)*nx    ]) 
#define t_xy(ix,iy)     (txy[ix + (iy)*(nx+1)])
#define divV(ix,iy)     (div[ix + (iy)*nx    ]) 

// --------------------------------------------------------------------- //
// Physics
const DAT Lx   = 10.0;
const DAT Ly   = 10.0;
const DAT k    = 1.0;
const DAT rho  = 1.0;
const DAT G    = 1.0;
// Numerics
#define BLOCK_X 32
#define BLOCK_Y 32
#define GRID_X  4
#define GRID_Y  4
const int nx   = BLOCK_X*GRID_X - OVERLENGTH_X;
const int ny   = BLOCK_Y*GRID_Y - OVERLENGTH_Y;
const int nt   = 200;
const int nmax = 10;
const DAT dx   = Lx/((DAT)nx);
const DAT dy   = Ly/((DAT)ny);
const DAT dt   = min(dx,dy)/sqrt(k/rho)/2.5/2.0;
// --------------------------------------------------------------------- //
void save_info(int me, const int nx, const int ny){
    FILE* fid;
    fid=fopen("0_nxy.inf" ,"w"); fprintf(fid,"%d %d %d", PRECIS, nx, ny); fclose(fid);
}
#define save_info() save_info(0, nx, ny);

void save_array(DAT* A, int nx, int ny, int me, const char A_name[]){
    char* fname; FILE* fid; asprintf(&fname, "%d_%s.res" , me, A_name);
    fid=fopen(fname, "wb"); fwrite(A, sizeof(DAT), (nx)*(ny), fid); fclose(fid); free(fname);
}
#define SaveArray(A,nx,ny,A_name) gather(A,nx,ny); save_array(A##_h,nx,ny,0,A_name);

void  clean_cuda(){ 
    hipError_t ce = hipGetLastError();
    if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset();}
}
// Timer
#include "sys/time.h"
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }
// MIN and MAX function //
DAT device_MAX=0.0;
#define NB_THREADS     (BLOCK_X*BLOCK_Y)
#define blockId        (blockIdx.x  +  blockIdx.y *gridDim.x)
#define threadId       (threadIdx.x + threadIdx.y*blockDim.x)
#define isBlockMaster  (threadIdx.x==0 && threadIdx.y==0)
// maxval //
#define block_max_init()  DAT __thread_maxval=0.0;
#define __thread_max(A,nx_A,ny_A)  if (iy<ny_A && ix<nx_A){ __thread_maxval = max((__thread_maxval) , (A[ix + iy*nx_A])); } 

__shared__ volatile  DAT __block_maxval;
#define __block_max(A,nx_A,ny_A)  __thread_max(A,nx_A,ny_A);  if (isBlockMaster){ __block_maxval=0; }  __syncthreads(); \
                                  for (int i=0; i < (NB_THREADS); i++){ if (i==threadId){ __block_maxval = max(__block_maxval,__thread_maxval); }  __syncthreads(); }

__global__ void __device_max_d(DAT*A, const int nx_A,const int ny_A, DAT*__device_maxval){
  block_max_init();
  for_ix for_iy
  // find the maxval for each block
  __block_max(A,nx_A,ny_A);
  __device_maxval[blockId] = __block_maxval;
}

#define __DEVICE_max(A,nx_A,ny_A)  __device_max_d<<<grid, block>>>(A##_d, nx_A,ny_A, __device_maxval_d); \
                                   gather(__device_maxval,grid.x,grid.y); device_MAX=(DAT)0.0;           \
                                   for (int i=0; i < (grid.x*grid.y); i++){                              \
                                      device_MAX = max(device_MAX,__device_maxval_h[i]);                 \
                                   }                                                                     \
                                   A##_MAX = (device_MAX);
// --------------------------------------------------------------------- //
// Computing physics kernels
__global__ void init(DAT* x, DAT* y, DAT* P, const DAT Lx, const DAT Ly, const DAT dx, const DAT dy, const int nx, const int ny){
    for_ix for_iy
    if (iy<ny && ix<nx){ x[ix + iy*nx] = (DAT)ix*dx + (-Lx+dx)/2.0; }
    if (iy<ny && ix<nx){ y[ix + iy*nx] = (DAT)iy*dy + (-Ly+dy)/2.0; }
    if (iy<ny && ix<nx){ P[ix + iy*nx] = exp(-(x[ix + iy*nx]*x[ix + iy*nx]) -(y[ix + iy*nx]*y[ix + iy*nx])); }
}
__global__ void compute_P(DAT* Vx, DAT* Vy, DAT* P, DAT* txx, DAT* tyy, DAT* txy, DAT* div, DAT one_dx, DAT one_dy, DAT dtk, DAT dt2G, DAT dtG, const int nx, const int ny){
    for_ix for_iy
    if (iy<ny && ix<nx){ divV(ix,iy) = one_dx*(Velx(ix+1,iy)-Velx(ix,iy)) + one_dy*(Vely(ix,iy+1)-Vely(ix,iy)); }
    if (iy<ny && ix<nx){ Pres(ix,iy) = Pres(ix,iy) - dtk*divV(ix,iy); }

    if (iy<ny && ix<nx){ t_xx(ix,iy) = t_xx(ix,iy) + dt2G*( one_dx*(Velx(ix+1,iy)-Velx(ix,iy)) - 0.33*divV(ix,iy)); }
    if (iy<ny && ix<nx){ t_yy(ix,iy) = t_yy(ix,iy) + dt2G*( one_dy*(Vely(ix,iy+1)-Vely(ix,iy)) - 0.33*divV(ix,iy)); }
    
    if (iy>0 && iy<ny && ix>0 && ix<nx){ t_xy(ix,iy) = t_xy(ix,iy) + dtG*(one_dy*(Velx(ix,iy)-Velx(ix,iy-1)) + one_dx*(Vely(ix,iy)-Vely(ix-1,iy))); }
}
__global__ void compute_V(DAT* Vx, DAT* Vy, DAT* P, DAT* txx, DAT* tyy, DAT* txy, DAT dt_dx_rho, DAT dt_dy_rho, const int nx, const int ny){
    for_ix for_iy
    if (iy<ny && ix>0 && ix<nx){ Velx(ix,iy) = Velx(ix,iy) - dt_dx_rho*(Pres(ix,iy)-Pres(ix-1,iy) - (t_xx(ix,iy)-t_xx(ix-1,iy))) + dt_dy_rho*(t_xy(ix,iy+1)-t_xy(ix,iy)); }
    if (iy>0 && iy<ny && ix<nx){ Vely(ix,iy) = Vely(ix,iy) - dt_dy_rho*(Pres(ix,iy)-Pres(ix,iy-1) - (t_yy(ix,iy)-t_yy(ix,iy-1))) + dt_dx_rho*(t_xy(ix+1,iy)-t_xy(ix,iy)); }
}
int main(){
    int i, it;
    size_t N=nx*ny, mem=N*sizeof(DAT);
    // Set up GPU
    int gpu_id=-1;
    dim3 grid, block;
    block.x = BLOCK_X; grid.x = GRID_X;
    block.y = BLOCK_Y; grid.y = GRID_Y;
    gpu_id = GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n",gpu_id); 
    printf("%dx%d, %1.3f GB, %d iterations.\n", nx,ny, 5*mem/1024./1024./1024., nt);
    printf("Launching (%dx%d) grid of (%dx%d) blocks.\n", grid.x, grid.y, block.x, block.y);
    // Initial arrays
    zeros(x   ,nx  ,ny  );
    zeros(y   ,nx  ,ny  );
    zeros(P   ,nx  ,ny  );
    zeros(Vx  ,nx+1,ny  );
    zeros(Vy  ,nx  ,ny+1);
    zeros(div ,nx  ,ny  );
    zeros(txx ,nx  ,ny  );
    zeros(tyy ,nx  ,ny  );
    zeros(txy ,nx+1,ny+1);
	zeros(__device_maxval ,grid.x,grid.y);
	DAT P_MAX = 0.0;
    // Initial conditions
    init<<<grid,block>>>(x_d, y_d, P_d, Lx, Ly, dx, dy, nx, ny);              hipDeviceSynchronize();
    DAT dt_dx_rho = dt/dx/rho;
    DAT dt_dy_rho = dt/dy/rho;
    DAT one_dx    = (DAT)1.0/dx;
    DAT one_dy    = (DAT)1.0/dy;
    DAT dtk       = dt*k;
    DAT dt2G      = dt*(DAT)2.0*G;
    DAT dtG       = dt*G;
    // Action
    for (it=0;it<nt;it++){
        if (it==3){ tic(); } 
        compute_P<<<grid,block>>>(Vx_d, Vy_d, P_d, txx_d, tyy_d, txy_d, div_d, one_dx, one_dy, dtk, dt2G, dtG, nx, ny);  hipDeviceSynchronize();
        compute_V<<<grid,block>>>(Vx_d, Vy_d, P_d, txx_d, tyy_d, txy_d, dt_dx_rho, dt_dy_rho,                  nx, ny);  hipDeviceSynchronize();
        if (it%nmax==0){ __DEVICE_max(P,nx,ny); printf("max(P)=%1.3e \n", P_MAX); }
    }//it
    tim("Time (s), Effective MTP (GB/s)", mem*(nt-3)*6*2/1024./1024./1024.);
    save_info();
    SaveArray(P ,nx  ,ny  ,"P" );
    SaveArray(Vx,nx+1,ny  ,"Vx");
    SaveArray(Vy,nx  ,ny+1,"Vy");
    free_all(x);
    free_all(y);
    free_all(P);
    free_all(Vx);
    free_all(Vy);
    free_all(div);
    free_all(txx);
    free_all(tyy);
    free_all(txy);

    clean_cuda();
}
