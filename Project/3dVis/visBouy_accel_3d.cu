#include "hip/hip_runtime.h"
// 3D Bouyant Ball viscous code
// nvcc -arch=sm_70 -O3 visBouy_accel_3d.cu
// run: ./a.out
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"

//#define USE_SINGLE_PRECISION    /* Comment this line using "//" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT     float
#define PRECIS  4
#else
#define DAT     double
#define PRECIS  8
#endif

#define GPU_ID        0
#define OVERLENGTH_X  1
#define OVERLENGTH_Y  1
#define OVERLENGTH_Z  1
        
#define zeros(A,nx,ny,nz)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc(((nx)*(ny)*(nz))*sizeof(DAT)); \
                        for(i=0; i < ((nx)*(ny)*(nz)); i++){ A##_h[i]=(DAT)0.0; }              \
                        hipMalloc(&A##_d      ,((nx)*(ny)*(nz))*sizeof(DAT));                 \
                        hipMemcpy( A##_d,A##_h,((nx)*(ny)*(nz))*sizeof(DAT),hipMemcpyHostToDevice);
#define free_all(A)     free(A##_h); hipFree(A##_d);
#define gather(A,nx,ny,nz) hipMemcpy( A##_h,A##_d,((nx)*(ny)*(nz))*sizeof(DAT),hipMemcpyDeviceToHost);
// --------------------------------------------------------------------- //
// Physics
const DAT Lx  = 10.0;
const DAT Ly  = 10.0;
const DAT Lz  = 10.0;
const DAT k   = 1.0;
const DAT rhoi= 10.0;
const DAT g   = -10.0;
const DAT eta = 1.0;
const DAT nu  = 6.0;
const DAT epsi= 1.0e-6;
// Numerics
#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_Z 8
#define GRID_X  4
#define GRID_Y  4 
#define GRID_Z  4 
const int nx = BLOCK_X*GRID_X - OVERLENGTH_X;
const int ny = BLOCK_Y*GRID_Y - OVERLENGTH_Y;
const int nz = BLOCK_Z*GRID_Z - OVERLENGTH_Z;
const int nt = 20000;
const int nmax = 100;
const DAT dx = Lx/((DAT)nx);
const DAT dy = Ly/((DAT)ny);
const DAT dz = Lz/((DAT)nz);
const DAT dtV = (min(dx, min(dy,dz))*min(dx,min(dy,dz)))/(eta*4.1*((DAT)4));
const DAT dtP = 4.1*eta/((DAT)(4*ny));
// --------------------------------------------------------------------- //
void save_info(int me, const int nx, const int ny, const int nz){
    FILE* fid;
    if (me==0){ fid=fopen("0_nxyz.inf" ,"w"); fprintf(fid,"%d %d %d %d", PRECIS, nx, ny, nz); fclose(fid); }
}
#define save_info() save_info(me, nx, ny, nz);

void save_array(DAT* A, int nx, int ny, int nz, int me, const char A_name[]){
    char* fname; FILE* fid; asprintf(&fname, "%d_%s.res" , me, A_name);
    fid=fopen(fname, "wb"); fwrite(A, sizeof(DAT), (nx)*(ny)*(nz), fid); fclose(fid); free(fname);
}
#define SaveArray(A,nx,ny,nz,A_name) gather(A,nx,ny,nz); save_array(A##_h,nx,ny,nz,me,A_name);

void  clean_cuda(){ 
    hipError_t ce = hipGetLastError();
    if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset();}
}
// Timer
#include "sys/time.h"
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }
// MIN and MAX function //
DAT device_MAX=0.0;
#define NB_THREADS     (BLOCK_X*BLOCK_Y*BLOCK_Z)
#define blockId        (blockIdx.x  +  blockIdx.y *gridDim.x + blockIdx.z*gridDim.y*gridDim.x)
#define threadId       (threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.y*blockDim.x)
#define isBlockMaster  (threadIdx.x==0 && threadIdx.y==0 && threadIdx.z ==0)
// maxval //
#define block_max_init()  DAT __thread_maxval=0.0;
#define __thread_max(A,nx_A,ny_A,nz_A)  if (iy<ny_A && ix<nx_A && iz<nz_A){ __thread_maxval = max((__thread_maxval) , abs(A[ix + iy*nx_A +iz*ny_A*nx_A])); } 

__shared__ volatile  DAT __block_maxval;
#define __block_max(A,nx_A,ny_A,nz_A)  __thread_max(A,nx_A,ny_A,nz_A);  if (isBlockMaster){ __block_maxval=0; }  __syncthreads(); \
                                  for (int i=0; i < (NB_THREADS); i++){ if (i==threadId){ __block_maxval = max(__block_maxval,__thread_maxval); }  __syncthreads(); }

__global__ void __device_max_d(DAT*A, const int nx_A,const int ny_A, const int nz_A, DAT*__device_maxval){
  block_max_init();
  int ix = blockIdx.x*blockDim.x + threadIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int iz = blockIdx.z*blockDim.z + threadIdx.z;
  // find the maxval for each block
  __block_max(A,nx_A,ny_A,nz_A);
  __device_maxval[blockId] = __block_maxval;
}

#define __DEVICE_max(A,nx_A,ny_A,nz_A)  __device_max_d<<<grid, block>>>(A##_d, nx_A, ny_A, nz_A, __device_maxval_d); \
                                   gather(__device_maxval,grid.x,grid.y,grid.z); device_MAX=(DAT)0.0;           \
                                   for (int i=0; i < (grid.x*grid.y*grid.z); i++){                              \
                                      device_MAX = max(device_MAX,__device_maxval_h[i]);                 \
                                   }                                                                     \
                                   A##_MAX = (device_MAX);

// --------------------------------------------------------------------- //
// Computing physics kernels
__global__ void init(DAT* x, DAT* y, DAT* z, DAT* rho, const DAT Lx, const DAT Ly, const DAT Lz, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny, const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx && iz<nz){ x[ix+iy*nx+iz*nx*ny] = (DAT)ix*dx + (-Lx+dx)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ y[ix+iy*nx+iz*nx*ny] = (DAT)iy*dy + (-Ly+dy)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ z[ix+iy*nx+iz*nx*ny] = (DAT)iz*dz + (-Lz+dz)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ 
        if(x[ix+iy*nx+iz*nx*ny]*x[ix+iy*nx+iz*nx*ny] + y[ix+iy*nx+iz*nx*ny]*y[ix+iy*nx+iz*nx*ny] + z[ix+iy*nx+iz*nx*ny]*z[ix+iy*nx+iz*nx*ny] < 1){
            rho[ix+iy*nx+iz*nx*ny]=rhoi;
        }
    }
}
__global__ void compute_V(DAT* Vx, DAT* Vy, DAT* Vz, DAT* P, DAT* Txx, DAT* Tyy, DAT* Tzz, DAT* Txy, DAT* Txz, DAT* Tyz, DAT* dVxdt, DAT* dVydt, DAT* dVzdt, 
                            DAT* Rx, DAT* Ry, DAT* Rz, DAT* rho, const DAT dtV, const DAT g, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny,  const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (ix>0 && iy<ny && ix<nx && iz<nz){
        Rx[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny] = 1 * (
            -1*(P[ ix +(iy  )* nx   +(iz  )* nx   * ny   ] -   P[(ix-1)+(iy  )* nx   +(iz  )* nx   * ny   ])/dx
           + (Txx[ ix +(iy  )* nx   +(iz  )* nx   * ny   ] - Txx[(ix-1)+(iy  )* nx   +(iz  )* nx   * ny   ])/dx
           + (Txy[(ix)+(iy+1)*(nx+1)+(iz  )*(nx+1)*(ny+1)] - Txy[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny+1)])/dy
           + (Txz[(ix)+(iy  )*(nx+1)+(iz+1)*(nx+1)*(ny  )] - Txz[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dz);
            dVxdt[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny] = (1-nu/nx)*dVxdt[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny] + Rx[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny];
            Vx[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny] = Vx[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny] + dtV*dVxdt[ix+(iy)*(nx+1)+(iz)*(nx+1)*ny];
    }
    if (iy>0 && iy<ny && ix<nx && iz<nz){
         Ry[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)] = 1 * (
                -1*(P[(ix  )+(iy  )* nx   +(iz  )* nx   * ny   ] -   P[(ix  )+(iy-1)* nx   +(iz  )* nx   * ny   ])/dy
               + (Tyy[(ix  )+(iy  )* nx   +(iz  )* nx   * ny   ] - Tyy[(ix  )+(iy-1)* nx   +(iz  )* nx   * ny   ])/dy
               + (Txy[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny+1)] - Txy[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny+1)])/dx
               + (Tyz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny+1)] - Tyz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dz
          + .5*g*(rho[(ix  )+(iy  )* nx   +(iz  )* nx   * ny   ] + rho[(ix  )+(iy-1)* nx   +(iz  )* nx   * ny   ]));
                dVydt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)] = (1-nu/ny)*dVydt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)] + Ry[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)];
                Vy[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)] = Vy[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)] + dtV*dVydt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny+1)];
    }
    if (iz>0 && iy<ny && ix<nx && iz<nz){
        Rz[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )] = 1 * (
            -1*(P[(ix  )+(iy  )* nx   +(iz  )* nx   * ny   ] -   P[(ix  )+(iy  )* nx   +(iz-1)* nx   * ny   ])/dz
           + (Tzz[(ix  )+(iy  )* nx   +(iz  )* nx   * ny   ] - Tzz[(ix  )+(iy  )* nx   +(iz-1)* nx   * ny   ])/dz
           + (Txz[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )] - Txz[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx
           + (Tyz[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)] - Tyz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy);
            dVzdt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )] = (1-nu/nz)*dVzdt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )] + Rz[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )];
            Vz[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )] = Vz[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )] + dtV*dVzdt[ix+(iy)*(nx  )+(iz)*(nx  )*(ny  )];
    }
}
__global__ void compute_P(DAT* Vx, DAT* Vy, DAT* Vz, DAT* P, const DAT dtP, const DAT k, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny, const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx && iz<nz){
        P[ix+(iy)*nx+(iz)*nx*ny] = P[ix+(iy)*nx+(iz)*nx*ny] - dtP*k*(
                  (Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx+
                  (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy+
                  (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz);
    }
}
__global__ void compute_T(DAT* Vx, DAT* Vy, DAT* Vz, DAT* Txx, DAT* Tyy, DAT* Tzz, DAT* Txy, DAT* Txz, DAT* Tyz, const DAT eta, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny, const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx && iz<nz){
        Txx[ix+(iy)*nx+(iz)*nx*ny] = 2*eta*(
              (Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx - 
             ((Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx+
              (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy+
              (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz)/((DAT)3));
        Tyy[ix+(iy)*nx+(iz)*nx*ny] = 2*eta*(
              (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy - 
             ((Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx+
              (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy+
              (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz)/((DAT)3));
        Tzz[ix+(iy)*nx+(iz)*nx*ny] = 2*eta*(
              (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz - 
             ((Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx+
              (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy+
              (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz)/((DAT)3));
    }
    if(iy<ny && ix<nx && iz<nz && ix>0  && iy >0){
        Txy[(ix)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny+1)] = eta*(
            (Vx[(ix)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )] - Vx[(ix  )+(iy-1)*(nx+1)+(iz  )*(nx+1)*(ny  )])/dy + 
            (Vy[(ix)+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)] - Vy[(ix-1)+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dx);
    }
    if(iy<ny && ix<nx && iz<nz && ix>0  && iz >0){
        Txz[(ix)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )] = eta*(
           (Vx[(ix)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )] - Vx[(ix  )+(iy  )*(nx+1)+(iz-1)*(nx+1)*(ny  )])/dz + 
           (Vz[(ix)+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )] - Vz[(ix-1)+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dx);
    }
    if(iy<ny && ix<nx && iz<nz && iy>0  && iz >0){
        Tyz[(ix)+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)] = eta*(
           (Vy[(ix)+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)] - Vy[(ix)+(iy  )*(nx  )+(iz-1)*(nx  )*(ny+1)])/dz + 
           (Vz[(ix)+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )] - Vz[(ix)+(iy-1)*(nx  )+(iz  )*(nx  )*(ny  )])/dy);
    }
}
int main(){
    int i, it;
    size_t N=nx*ny*nz, mem=N*sizeof(DAT);
    // Set up GPU
    int gpu_id=-1;
    int me = 0;
    dim3 grid, block;
    block.x = BLOCK_X; grid.x = GRID_X;
    block.y = BLOCK_Y; grid.y = GRID_Y;
    block.z = BLOCK_Z; grid.z = GRID_Z;
    gpu_id = GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n",gpu_id);
    printf("%dx%dx%d, %1.3f GB, %d iterations.\n", nx,ny,nz, 20*mem/1024./1024./1024., nt);
    printf("Launching (%dx%dx%d) grid of (%dx%dx%d) blocks.\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
    // Initial arrays
    zeros(x    ,nx  ,ny  ,nz  );
    zeros(y    ,nx  ,ny  ,nz  );
    zeros(z    ,nx  ,ny  ,nz  );
    zeros(rho  ,nx  ,ny  ,nz  );
    zeros(P    ,nx  ,ny  ,nz  );
    zeros(Txx  ,nx  ,ny  ,nz  );
    zeros(Tyy  ,nx  ,ny  ,nz  );
    zeros(Tzz  ,nx  ,ny  ,nz  );
    zeros(Txy  ,nx+1,ny+1,nz  );
    zeros(Txz  ,nx+1,ny  ,nz+1);
    zeros(Tyz  ,nx  ,ny+1,nz+1);
    zeros(Vx   ,nx+1,ny  ,nz  );
    zeros(Vy   ,nx  ,ny+1,nz  );
    zeros(Vz   ,nx+1,ny  ,nz+1);
    zeros(dVxdt,nx+1,ny  ,nz  );
    zeros(dVydt,nx  ,ny+1,nz  );
    zeros(dVzdt,nx+1,ny  ,nz+1);
    zeros(Rx   ,nx+1,ny  ,nz  );
    zeros(Ry   ,nx  ,ny+1,nz  );
    zeros(Rz   ,nx+1,ny  ,nz+1);
    zeros(__device_maxval ,grid.x,grid.y,grid.z);
    DAT Rx_MAX = 1.0;
    DAT Ry_MAX = 0.0;
    DAT Rz_MAX = 0.0;
    // Initial conditions
    init<<<grid,block>>>(x_d, y_d, z_d, rho_d, Lx, Ly, Lz, dx, dy, dz, nx, ny, nz);              hipDeviceSynchronize();
    // Action
    for (it=0;it<nt;it++){
        if (it==1){ tic(); } 
        compute_P<<<grid,block>>>(Vx_d, Vy_d, Vz_d, P_d, dtP, k, dx, dy, dz, nx, ny, nz);  hipDeviceSynchronize();
        compute_T<<<grid,block>>>(Vx_d, Vy_d, Vz_d, Txx_d, Tyy_d, Tzz_d, Txy_d, Txz_d, Tyz_d, eta, dx, dy, dz, nx, ny, nz);  hipDeviceSynchronize();
        compute_V<<<grid,block>>>(Vx_d, Vy_d, Vz_d, P_d, Txx_d, Tyy_d, Tzz_d, Txy_d, Txz_d, Tyz_d, dVxdt_d, dVydt_d, dVzdt_d,
                                  Rx_d, Ry_d, Rz_d, rho_d, dtV, g, dx ,dy ,dz ,nx ,ny ,nz );  hipDeviceSynchronize();
        __DEVICE_max(Rx,nx,ny,nz);
        __DEVICE_max(Ry,nx,ny,nz);
        __DEVICE_max(Rz,nx,ny,nz);
        if (it%nmax==0){ printf("max(Rx,Ry,Rz)=%1.3e, %1.3e, %1.3e \n", Rx_MAX, Ry_MAX, Rz_MAX); }
        if (Rx_MAX < epsi && Ry_MAX < epsi && Rz_MAX < epsi && it > nmax){
            printf("Broke on iteration %d \n",it);
            printf("max(Rx,Ry,Rz)=%1.3e, %1.3e, %1.3e \n", Rx_MAX, Ry_MAX, Rz_MAX);
            break;
        }
    }//it
    tim("Time (s), Effective MTP (GB/s)", mem*(nt-3)*20/1024./1024./1024.);
    save_info();
    SaveArray(P ,nx  ,ny  ,nz  ,"P" );
    SaveArray(Vx,nx+1,ny  ,nz  ,"Vx");
    SaveArray(Vy,nx  ,ny+1,nz  ,"Vy");
    SaveArray(Vz,nx  ,ny  ,nz+1,"Vz");
    SaveArray(Rx,nx+1,ny  ,nz  ,"Rx");
    SaveArray(Ry,nx  ,ny+1,nz  ,"Ry");
    SaveArray(Rz,nx  ,ny  ,nz+1,"Rz");
    SaveArray(Txx,nx  ,ny  ,nz  ,"Txx");
    SaveArray(Tyy,nx  ,ny  ,nz  ,"Tyy");
    SaveArray(Tzz,nx  ,ny  ,nz  ,"Tzz");
    SaveArray(Txy,nx+1,ny+1,nz  ,"Txy");
    SaveArray(Txz,nx+1,ny  ,nz+1,"Txz");
    SaveArray(Tyz,nx  ,ny+1,nz+1,"Tyz");
    free_all(x );
    free_all(y );
    free_all(z );
    free_all(rho);
    free_all(P );
    free_all(Vx);
    free_all(Vy);
    free_all(Vz);
    free_all(dVxdt);
    free_all(dVydt);
    free_all(dVzdt);
    free_all(Rx);
    free_all(Ry);
    free_all(Rz);
	free_all(Txx);
    free_all(Tyy);
    free_all(Tzz);
    free_all(Txy);
    free_all(Txz);
    free_all(Tyz);
    clean_cuda();
}
