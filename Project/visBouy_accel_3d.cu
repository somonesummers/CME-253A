#include "hip/hip_runtime.h"
// 3D Bouyant Ball viscous code
// nvcc -arch=sm_70 -O3 wave_2D_Vis_v2.cu
// run: ./a.out
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "hip/hip_runtime.h"

//#define USE_SINGLE_PRECISION    /* Comment this line using "//" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT     float
#define PRECIS  4
#else
#define DAT     double
#define PRECIS  8
#endif

#define GPU_ID        0
#define OVERLENGTH_X  1
#define OVERLENGTH_Y  1
#define OVERLENGTH_Z  1
        
#define zeros(A,nx,ny,nz)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc(((nx)*(ny)*(nz))*sizeof(DAT)); \
                        for(i=0; i < ((nx)*(ny)*(nz)); i++){ A##_h[i]=(DAT)0.0; }              \
                        hipMalloc(&A##_d      ,((nx)*(ny)*(nz))*sizeof(DAT));                 \
                        hipMemcpy( A##_d,A##_h,((nx)*(ny)*(nz))*sizeof(DAT),hipMemcpyHostToDevice);
#define free_all(A)     free(A##_h); hipFree(A##_d);
#define gather(A,nx,ny,nz) hipMemcpy( A##_h,A##_d,((nx)*(ny)*(nz))*sizeof(DAT),hipMemcpyDeviceToHost);
// --------------------------------------------------------------------- //
// Physics
const DAT Lx  = 10.0;
const DAT Ly  = 10.0;
const DAT Lz  = 10.0;
const DAT k   = 1.0;
const DAT rhoi= 10.0;
const DAT eta = 1.0;
const DAT nu  = 6.0;
const DAT epsi= 1.0e-6;
// Numerics
#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_Z 8
#define GRID_X  4
#define GRID_Y  4 
#define GRID_z  4 
const int nx = BLOCK_X*GRID_X - OVERLENGTH_X;
const int ny = BLOCK_Y*GRID_Y - OVERLENGTH_Y;
const int nz = BLOCK_Z*GRID_Z - OVERLENGTH_Z;
const int nt = 40000;
const DAT dx = Lx/((DAT)nx);
const DAT dy = Ly/((DAT)ny);
const DAT dz = Lz/((DAT)nz);
const DAT dtV = (min(dx,dy,dz)*min(dx,dy,dz))/(eta*4.1*((DAT)4));
const DAT dtP = 4.1*eta/((DAT)(4*ny));
// --------------------------------------------------------------------- //
void save_info(int me, const int nx, const int ny){
    FILE* fid;
    if (me==0){ fid=fopen("0_nxy.inf" ,"w"); fprintf(fid,"%d %d %d", PRECIS, nx, ny); fclose(fid); }
}
#define save_info() save_info(me, nx, ny);

void save_array(DAT* A, int nx, int ny, int me, const char A_name[]){
    char* fname; FILE* fid; asprintf(&fname, "%d_%s.res" , me, A_name);
    fid=fopen(fname, "wb"); fwrite(A, sizeof(DAT), (nx)*(ny), fid); fclose(fid); free(fname);
}
#define SaveArray(A,nx,ny,A_name) gather(A,nx,ny); save_array(A##_h,nx,ny,me,A_name);

void  clean_cuda(){ 
    hipError_t ce = hipGetLastError();
    if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset();}
}
// Timer
#include "sys/time.h"
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }
// --------------------------------------------------------------------- //
// Computing physics kernels
__global__ void init(DAT* x, DAT* y, DAT* z, DAT* rho, const DAT Lx, const DAT Ly, const DAT Lz, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny, const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx && iz<nz){ x[ix+iy*nx+iz*nx*ny] = (DAT)ix*dx + (-Lx+dx)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ y[ix+iy*nx+iz*nx*ny] = (DAT)iy*dy + (-Ly+dy)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ z[ix+iy*nx+iz*nx*ny] = (DAT)iz*dz + (-Lz+dz)/2.0; }
    if (iy<ny && ix<nx && iz<nz){ 
        if(x[ix+iy*nx+iz*nx*ny]*x[ix+iy*nx+iz*nx*ny] + y[ix+iy*nx+iz*nx*ny]*y[ix+iy*nx+iz*nx*ny] + z[ix+iy*nx+iz*nx*ny]*z[ix+iy*nx+iz*nx*ny] < 1){
            rho[ix+iy*nx+iz*nx*ny]=rhoi;
        }
    }
}
__global__ void compute_V(DAT* Vx, DAT* Vy, DAT* P, DAT* Txx, DAT* Tyy, DAT* Txy, const DAT dt, const DAT rho, const DAT dx, const DAT dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix>0 && ix<nx){
        Vx[ix+(iy)*(nx+1)] = Vx[ix+(iy)*(nx+1)] + dt/rho*(
                -1*(P[ix+(iy)*nx]-P[(ix-1)+(iy)*nx])/dx
                + (Txx[ix+(iy)*nx] - Txx[(ix-1)+(iy)*nx])/dx
                + (Txy[ix+(iy+1)*(nx+1)] - Txy[ix+(iy)*(nx+1)])/dy);
    }
    if (iy>0 && iy<ny && ix<nx){
        Vy[ix+(iy)*(nx)] = Vy[ix+(iy)*(nx)] + dt/rho*(
                -1*(P[ix+(iy)*nx]-P[ix+(iy-1)*nx])/dy
                + (Tyy[ix+(iy)*nx] - Tyy[ix+(iy-1)*nx])/dy
                + (Txy[(ix+1)+(iy)*(nx+1)] - Txy[ix+(iy)*(nx+1)])/dx);
    }
}
__global__ void compute_P(DAT* Vx, DAT* Vy, DAT* Vz, DAT* P, const DAT dt, const DAT k, const DAT dx, const DAT dy, const DAT dz, const int nx, const int ny, const int nz){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx && iz<nz){
        P[ix+(iy)*nx+(iz)*nx*ny] = P[ix+(iy)*nx+(iz)*nx*ny] - dtP*k*(...
                  (Vx[(ix+1)+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )]-Vx[(ix  )+(iy  )*(nx+1)+(iz  )*(nx+1)*(ny  )])/dx+...
                  (Vy[(ix  )+(iy+1)*(nx  )+(iz  )*(nx  )*(ny+1)]-Vy[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny+1)])/dy+...
                  (Vz[(ix  )+(iy  )*(nx  )+(iz+1)*(nx  )*(ny  )]-Vz[(ix  )+(iy  )*(nx  )+(iz  )*(nx  )*(ny  )])/dz);
    }
}
__global__ void compute_T(DAT* Vx, DAT* Vy, DAT* P, DAT* Txx, DAT* Tyy, DAT* Txy, const DAT mu, const DAT dt, const DAT dx, const DAT dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x; // thread ID, dimension x
    int iy = blockIdx.y*blockDim.y + threadIdx.y; // thread ID, dimension y
    int iz = blockIdx.z*blockDim.z + threadIdx.z; // thread ID, dimension z
    if (iy<ny && ix<nx){
        Txx[ix+(iy)*nx] = 2*mu*(
                         (Vx[(ix+1)+(iy  )*(nx+1)]-Vx[ix+(iy)*(nx+1)])/dx - 
                        ((Vx[(ix+1)+(iy  )*(nx+1)]-Vx[ix+(iy)*(nx+1)])/dx +
                         (Vy[ ix   +(iy+1)*(nx  )]-Vy[ix+(iy)*(nx  )])/dy)/((DAT)3));
        Tyy[ix+(iy)*nx] = 2*mu*(
                         (Vy[ ix   +(iy+1)*(nx  )]-Vy[ix+(iy)*(nx  )])/dy - 
                        ((Vx[(ix+1)+(iy  )*(nx+1)]-Vx[ix+(iy)*(nx+1)])/dx +
                         (Vy[ ix   +(iy+1)*(nx  )]-Vy[ix+(iy)*(nx  )])/dy)/((DAT)3));
    }
    if(iy<ny && ix<nx && ix>0  && iy >0){
        Txy[ix+(iy)*(nx+1)] = mu*(
                   (Vx[ix+(iy)*(nx+1)] - Vx[ ix   +(iy-1)*(nx+1)])/dy + 
                   (Vy[ix+(iy)*(nx  )] - Vy[(ix-1)+(iy  )*(nx  )])/dx);
    }
}
int main(){
    int i, it;
    size_t N=nx*ny*nz, mem=N*sizeof(DAT);
    // Set up GPU
    int gpu_id=-1;
    int me = 0;
    dim3 grid, block;
    block.x = BLOCK_X; grid.x = GRID_X;
    block.y = BLOCK_Y; grid.y = GRID_Y;
    block.z = BLOCK_Z; grid.z = GRID_Z;
    gpu_id = GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n",gpu_id);
    printf("%dx%d, %1.3f GB, %d iterations.\n", nx,ny, 5*mem/1024./1024./1024., nt);
    printf("Launching (%dx%dx%d) grid of (%dx%dx%d) blocks.\n", grid.x, grid.y, grid.z, block.x, block.y, block.z);
    // Initial arrays
    zeros(x    ,nx  ,ny  ,nz  );
    zeros(y    ,nx  ,ny  ,nz  );
    zeros(z    ,nx  ,ny  ,nz  );
    zeros(rho  ,nx  ,ny  ,nz  );
    zeros(P    ,nx  ,ny  ,nz  );
    zeros(Txx  ,nx  ,ny  ,nz  );
    zeros(Tyy  ,nx  ,ny  ,nz  );
    zeros(Tzz  ,nx  ,ny  ,nz  );
    zeros(Txy  ,nx+1,ny+1,nz  );
    zeros(Txz  ,nx+1,ny  ,nz+1);
    zeros(Tyz  ,nx  ,ny+1,nz+1);
    zeros(Vx   ,nx+1,ny  ,nz  );
    zeros(Vy   ,nx  ,ny+1,nz  );
    zeros(Vz   ,nx+1,ny  ,nz+1);
    zeros(dVxdt,nx+1,ny  ,nz  );
    zeros(dVydt,nx  ,ny+1,nz  );
    zeros(dVzdt,nx+1,ny  ,nz+1);
    zeros(Rx   ,nx+1,ny  ,nz  );
    zeros(Ry   ,nx  ,ny+1,nz  );
    zeros(Rz   ,nx+1,ny  ,nz+1);
    // Initial conditions
    init<<<grid,block>>>(x_d, y_d, z_d, rho_d, Lx, Ly, Lz, dx, dy, dz, nx, ny, nz);              hipDeviceSynchronize();
    // Action
    for (it=0;it<nt;it++){
        if (it==1){ tic(); } 
        compute_V<<<grid,block>>>(Vx_d, Vy_d, P_d, Txx_d, Tyy_d, Txy_d, dt, rho, dx, dy, nx, ny);  hipDeviceSynchronize();
        compute_P<<<grid,block>>>(Vx_d, Vy_d, P_d, dt, k, dx, dy, nx, ny);  hipDeviceSynchronize();
        compute_T<<<grid,block>>>(Vx_d, Vy_d, P_d, Txx_d, Tyy_d, Txy_d, mu, dt, dx, dy, nx, ny);  hipDeviceSynchronize();
    }//it
    tim("Time (s), Effective MTP (GB/s)", mem*(nt-3)*4/1024./1024./1024.);
    save_info();
    SaveArray(P ,nx  ,ny  ,"P" );
    SaveArray(Vx,nx+1,ny  ,"Vx");
    SaveArray(Vy,nx  ,ny+1,"Vy");
    SaveArray(Txx,nx  ,ny  ,"Txx");
    SaveArray(Tyy,nx  ,ny  ,"Tyy");
    SaveArray(Txy,nx+1,ny+1,"Txy");
    free_all(x );
    free_all(y );
    free_all(P );
    free_all(Vx);
    free_all(Vy);
	free_all(Txx);
    free_all(Tyy);
    free_all(Txy);
    clean_cuda();
}
